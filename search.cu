/*
SPECIFICATIONS
GPU Memory 1GB DDR3
Memory Interface 128-bit
Memory Bandwidth 29.0GB/s
CUDA Cores 192
Graphics APIs Shader Model 5.0, OpenGL 4.4, DirectX 11
Compute APIs CUDA, DirectCompute,OpenCL
*/

/*
Build
nvcc -o teste.out teste.cu

Teste
sudo nvprof ./teste.bin
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
using namespace std;


int main(void) {
  // Estrutura
  // Tamanho
  // Dado


  string data[4][3];

  // dados
  data[0][0] = "test";
  data[1][0] = "casa";
  data[2][0] = "gato";
  data[3][0] = "foto";
  // size
  data[0][1] = "4";
  data[1][1] = "4";
  data[2][1] = "4";
  data[3][1] = "4";
  // hash sum
  data[0][2] = "4";
  data[1][2] = "4";
  data[2][2] = "4";
  data[3][2] = "4";

  string alvo = "gato";

  for(int i = 0; i < 4;i++){
    // pegando tamanho do dado ta tabela
    unsigned int sizeData = atoi(data[i][1].c_str());
    // Verificando o tamanho dos dados
    if(alvo.size() == sizeData){
      // Verificação linear dos dados
      for(int ind = 0; ind < sizeData; ){
        if(alvo.at(ind) == data[i][0].at(ind)){
          if(ind < sizeData - 1){
            printf("%s \n", data[i][0].c_str());
          }
          i++;
        } else {
          ind = sizeData;
        }
      } 
    }
  }
  return 0;
}
